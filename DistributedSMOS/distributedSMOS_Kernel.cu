#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <thrust/binary_search.h>
#include <thrust/device_vector.h>
#include <thrust/functional.h>
#include <thrust/random.h>
#include <thrust/sort.h>
#include <thrust/scan.h>
#include <thrust/execution_policy.h>
#include <thrust/transform_reduce.h>
#include <limits>
#include <math.h>
#include <sys/time.h>
#include <time.h>

#include "distributedSMOS_Kernel.cuh"

#define MAX_THREADS_PER_BLOCK 1024

/// ***********************************************************
/// ***********************************************************
/// **** Thrust Functions Library
/// ***********************************************************
/// ***********************************************************

using namespace std;

// thrust::minmax_element function
template <typename T>
void minmax_element_CALL(T* d_vector, int length, T* maximum, T* minimum) {
	thrust::device_ptr<T>dev_ptr(d_vector);
	thrust::pair<thrust::device_ptr<T>, thrust::device_ptr<T> > result =
                thrust::minmax_element(dev_ptr, dev_ptr + length);
	
	*minimum = *result.first;
	*maximum = *result.second;
}



template void minmax_element_CALL(int* d_vector, int length, int* maximum, int* minimum);
template void minmax_element_CALL(unsigned int* d_vector, int length, unsigned int* maximum, unsigned int* minimum);
template void minmax_element_CALL(double* d_vector, int length, double* maximum, double* minimum);
template void minmax_element_CALL(float* d_vector, int length, float* maximum, float* minimum);


// thrust::sort_by_key function
void sort_by_key_CALL(unsigned int* d_kVals, unsigned int* d_kIndices, int numKs) {
	thrust::device_ptr<unsigned int>kVals_ptr(d_kVals);
	thrust::device_ptr<unsigned int>kIndices_ptr(d_kIndices);
	thrust::sort_by_key(kVals_ptr, kVals_ptr + numKs, kIndices_ptr);
}



/// ***********************************************************
/// ***********************************************************
/// **** HELPER CPU FUNCTIONS
/// ***********************************************************
/// ***********************************************************

/* This function initializes a vector to all zeros on the host (CPU).
 */
template<typename T>
void setToAllZero (T * d_vector, int length) {
    hipMemset(d_vector, 0, length * sizeof(T));
}


/* This function finds the bin containing the kth element we are looking for (works on
   the host). While doing the scan, it stores the sum-so-far of the number of elements in
   the buckets where k values fall into.

   markedBuckets : buckets containing the corresponding k values
   sums : sum-so-far of the number of elements in the buckets where k values fall into
*/
int findKBuckets(unsigned int * h_bucketCount, int numBuckets, 
				const unsigned int * kVals, int numKs, unsigned int * sums, 
				unsigned int * markedBuckets, int numBlocks) {
    int kBucket = 0;
    int k;
    int sum = h_bucketCount[0];

    for(int i = 0; i < numKs; i++) {
        k = kVals[i];
        while ((sum < k) & (kBucket < numBuckets - 1)) {
            kBucket++;
            sum += h_bucketCount[kBucket];
        }
        markedBuckets[i] = kBucket;
        sums[i] = sum - h_bucketCount[kBucket];
    }

    return 0;
}
/*
 * This function updates the correct kth orderstats if the bin only contains one element. While going through the
 * list of orderstats, it updates K since we have reduced the problem size to elements in the kth bucket. In
 * addition, it updates the unique buckets list to avoid the situation where two order share the same buckets.
 *
 * kthBucketScanner:  sum-so-far of the number of elements in the buckets where k values fall into
 * uniqueBuckets:  the list to store all buckets which are active with no repeats
 */
template <typename T>
int updatekVals
	(unsigned int * kVals, int * numKs, T * output, unsigned int * kIndicies,
             int * length, int * lengthOld, unsigned int * h_bucketCount, unsigned int * markedBuckets,
             unsigned int * kthBucketScanner, unsigned int * reindexCounter,
             unsigned int * uniqueBuckets, unsigned int * uniqueBucketCounts,
             int * numUniqueBuckets, int * numUniqueBucketsOld) {
    int index = 0;
    int numKsindex = 0;
    *numUniqueBucketsOld = *numUniqueBuckets;
    *numUniqueBuckets = 0;
    *lengthOld = *length;

    // get the index of the first buckets with more than one elements in it
    // add the number of elements and updates correct kth order
    uniqueBuckets[0] = markedBuckets[index];
    uniqueBucketCounts[0] = h_bucketCount[markedBuckets[index]];
    reindexCounter[0] = 0;
    *numUniqueBuckets = 1;
    kVals[0] = kVals[index] - kthBucketScanner[index];
    kIndicies[0] = kIndicies[index];
    numKsindex++;
    index++;

    // go through the markedbuckets list. If there is more than one, updates it to uniqueBucket
    for ( ; index < *numKs; index++) {
        // case if the there is more than one element in the bucket and the bucket is not repeat with last one
        if (markedBuckets[index] != uniqueBuckets[(*numUniqueBuckets) - 1]) {
            uniqueBuckets[*numUniqueBuckets] = markedBuckets[index];
            uniqueBucketCounts[*numUniqueBuckets] = h_bucketCount[markedBuckets[index]];
            reindexCounter[*numUniqueBuckets] = reindexCounter[(*numUniqueBuckets) - 1]
                                                + uniqueBucketCounts[(*numUniqueBuckets) - 1];
            (*numUniqueBuckets)++;
        }

        // update korder
        kVals[numKsindex] = reindexCounter[(*numUniqueBuckets) - 1] + kVals[index] - kthBucketScanner[index];
        kIndicies[numKsindex] = kIndicies[index];
        numKsindex++;
    }

    // update numKs and length of vector
    *numKs = numKsindex;
    if (*numKs > 0)
        *length = reindexCounter[(*numUniqueBuckets) - 1] + uniqueBucketCounts[(*numUniqueBuckets) - 1];


    return 0;
}
template int updatekVals
			(unsigned int * kVals, int * numKs, int * output, unsigned int * kIndicies,
             int * length, int * lengthOld, unsigned int * h_bucketCount, unsigned int * markedBuckets,
             unsigned int * kthBucketScanner, unsigned int * reindexCounter,
             unsigned int * uniqueBuckets, unsigned int * uniqueBucketCounts,
             int * numUniqueBuckets, int * numUniqueBucketsOld);
template int updatekVals
			(unsigned int * kVals, int * numKs, unsigned int * output, unsigned int * kIndicies,
             int * length, int * lengthOld, unsigned int * h_bucketCount, unsigned int * markedBuckets,
             unsigned int * kthBucketScanner, unsigned int * reindexCounter,
             unsigned int * uniqueBuckets, unsigned int * uniqueBucketCounts,
             int * numUniqueBuckets, int * numUniqueBucketsOld);
template int updatekVals
			(unsigned int * kVals, int * numKs, float * output, unsigned int * kIndicies,
             int * length, int * lengthOld, unsigned int * h_bucketCount, unsigned int * markedBuckets,
             unsigned int * kthBucketScanner, unsigned int * reindexCounter,
             unsigned int * uniqueBuckets, unsigned int * uniqueBucketCounts,
             int * numUniqueBuckets, int * numUniqueBucketsOld);
template int updatekVals
			(unsigned int * kVals, int * numKs, double * output, unsigned int * kIndicies,
             int * length, int * lengthOld, unsigned int * h_bucketCount, unsigned int * markedBuckets,
             unsigned int * kthBucketScanner, unsigned int * reindexCounter,
             unsigned int * uniqueBuckets, unsigned int * uniqueBucketCounts,
             int * numUniqueBuckets, int * numUniqueBucketsOld);
             

/*
 * This function updates the correct kth orderstats if the bin only contains one element. While going through the
 * list of orderstats, it updates K since we have reduced the problem size to elements in the kth bucket. In
 * addition, it updates the unique buckets list to avoid the situation where two order share the same buckets.
 *
 * kthBucketScanner:  sum-so-far of the number of elements in the buckets where k values fall into
 * uniqueBuckets:  the list to store all buckets which are active with no repeats
 * tempKorderBucket:  buckets which have only one element. That is, the bucket with correct kth orderstats
 */
template <typename T>
int updatekVals_distributive
			(unsigned int * kVals, int * numKs, T * output, unsigned int * kIndicies,
             int * length, int * lengthOld, unsigned int * h_bucketCount, unsigned int * markedBuckets,
             unsigned int * kthBucketScanner, unsigned int * reindexCounter,
             unsigned int * uniqueBuckets, unsigned int * uniqueBucketCounts,
             int * numUniqueBuckets, int * numUniqueBucketsOld,
             unsigned int * tempKorderBucket, unsigned int * tempKorderIndeces, int * tempKorderLength) {
    int index = 0;
    int numKsindex = 0;
    *numUniqueBucketsOld = *numUniqueBuckets;
    *numUniqueBuckets = 0;
    *lengthOld = *length;
    *tempKorderLength = 0;

    // go through the markedbucket list. If there is only one element in array, we update it to tempKorderBucket
    while (index < *numKs) {
        if (h_bucketCount[markedBuckets[index]] == 1) {
            tempKorderIndeces[*tempKorderLength] = kIndicies[index];
            tempKorderBucket[*tempKorderLength] = markedBuckets[index];
            (*tempKorderLength)++;
            index++;
            continue;
        }

        break;
    }

    // get the index of the first buckets with more than one elements in it
    // add the number of elements and updates correct kth order
    if (index < *numKs) {
        uniqueBuckets[0] = markedBuckets[index];
        uniqueBucketCounts[0] = h_bucketCount[markedBuckets[index]];
        reindexCounter[0] = 0;
        *numUniqueBuckets = 1;
        kVals[0] = kVals[index] - kthBucketScanner[index];
        kIndicies[0] = kIndicies[index];
        numKsindex++;
        index++;
    }

    // go through the markedbuckets list. If there is only one element in that bucket, updates it to
    // tempKorderBucket; if there is more than one, updates it to uniqueBucket
    for ( ; index < *numKs; index++) {

        // case if there is only one element
        if (h_bucketCount[markedBuckets[index]] == 1) {
            tempKorderIndeces[*tempKorderLength] = kIndicies[index];
            tempKorderBucket[*tempKorderLength] = markedBuckets[index];
            (*tempKorderLength)++;
            continue;
        }

        // case if the there is more than one element in the bucket and the bucket is not repeat with last one
        if (markedBuckets[index] != uniqueBuckets[(*numUniqueBuckets) - 1]) {
            uniqueBuckets[*numUniqueBuckets] = markedBuckets[index];
            uniqueBucketCounts[*numUniqueBuckets] = h_bucketCount[markedBuckets[index]];
            reindexCounter[*numUniqueBuckets] = reindexCounter[(*numUniqueBuckets) - 1]
                                                + uniqueBucketCounts[(*numUniqueBuckets) - 1];
            (*numUniqueBuckets)++;
        }

        // update korder
        kVals[numKsindex] = reindexCounter[(*numUniqueBuckets) - 1] + kVals[index] - kthBucketScanner[index];
        kIndicies[numKsindex] = kIndicies[index];
        numKsindex++;
    }

    // update numKs and length of vector
    *numKs = numKsindex;
    if (*numKs > 0)
        *length = reindexCounter[(*numUniqueBuckets) - 1] + uniqueBucketCounts[(*numUniqueBuckets) - 1];


    return 0;
}


template int updatekVals_distributive
			(unsigned int * kVals, int * numKs, int * output, unsigned int * kIndicies,
             int * length, int * lengthOld, unsigned int * h_bucketCount, unsigned int * markedBuckets,
             unsigned int * kthBucketScanner, unsigned int * reindexCounter,
             unsigned int * uniqueBuckets, unsigned int * uniqueBucketCounts,
             int * numUniqueBuckets, int * numUniqueBucketsOld,
             unsigned int * tempKorderBucket, unsigned int * tempKorderIndeces, int * tempKorderLength);
template int updatekVals_distributive
			(unsigned int * kVals, int * numKs, unsigned int * output, unsigned int * kIndicies,
             int * length, int * lengthOld, unsigned int * h_bucketCount, unsigned int * markedBuckets,
             unsigned int * kthBucketScanner, unsigned int * reindexCounter,
             unsigned int * uniqueBuckets, unsigned int * uniqueBucketCounts,
             int * numUniqueBuckets, int * numUniqueBucketsOld,
             unsigned int * tempKorderBucket, unsigned int * tempKorderIndeces, int * tempKorderLength);
template int updatekVals_distributive
			(unsigned int * kVals, int * numKs, float * output, unsigned int * kIndicies,
             int * length, int * lengthOld, unsigned int * h_bucketCount, unsigned int * markedBuckets,
             unsigned int * kthBucketScanner, unsigned int * reindexCounter,
             unsigned int * uniqueBuckets, unsigned int * uniqueBucketCounts,
             int * numUniqueBuckets, int * numUniqueBucketsOld,
             unsigned int * tempKorderBucket, unsigned int * tempKorderIndeces, int * tempKorderLength);
template int updatekVals_distributive
			(unsigned int * kVals, int * numKs, double * output, unsigned int * kIndicies,
             int * length, int * lengthOld, unsigned int * h_bucketCount, unsigned int * markedBuckets,
             unsigned int * kthBucketScanner, unsigned int * reindexCounter,
             unsigned int * uniqueBuckets, unsigned int * uniqueBucketCounts,
             int * numUniqueBuckets, int * numUniqueBucketsOld,
             unsigned int * tempKorderBucket, unsigned int * tempKorderIndeces, int * tempKorderLength);
           
           
/*
 * Documentation
 * 
 */             
int updateReindexCounter_distributive
			(unsigned int* reindexCounter, unsigned int* h_bucketCount, unsigned int* uniqueBuckets,
			 int* length, int* length_Old, int numUniqueBuckets) {
	reindexCounter[0] = 0;
	*length_Old = *length;
	
	for (int i = 1; i < numUniqueBuckets; i++) {
		reindexCounter[i] = reindexCounter[i - 1] + h_bucketCount[uniqueBuckets[i - 1]];
	}
	
	*length = reindexCounter[numUniqueBuckets - 1] + 
			  h_bucketCount[uniqueBuckets[numUniqueBuckets - 1]];			 
	
	return 0;	 
}


/*
 * This function swap pointers for the two lists
 */
template <typename T>
void swapPointers(T** a, T** b) {
    T * temp = * a;
    * a = * b;
    * b = temp;
}

template void swapPointers(int** a, int** b);
template void swapPointers(unsigned int** a, unsigned int** b);
template void swapPointers(float** a, float** b);
template void swapPointers(double** a, double** b);

/*
 * Documentation
 */
template <typename T>
T absolute(T a) {
	if (a > 0.0)
		return a;
	else
		return -a;
}

template int absolute(int a);
template unsigned int absolute(unsigned int a);
template float absolute(float a);
template double absolute(double a); 


/// ***********************************************************
/// ***********************************************************
/// **** HELPER GPU FUNCTIONS-KERNELS
/// ***********************************************************
/// ***********************************************************

/*
 * Documentation
 */
template <typename T>
__global__ void generateSamples_distributive
					(T* d_vector, T* d_sampleVector, int length_local, int sampleSize_local, int offset) {
	
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int width = length_local / sampleSize_local;
	
	if (index < sampleSize_local) {
		for (int i = index; i < sampleSize_local; i += offset) {
			d_sampleVector[i] = d_vector[i * width];
		}
	}
}

/*
 * This function generate new buckets offset and slopes by giving the new pivots and number of elements in
 * that buckets
 *
 * pivotsLeft & pivotsRight:  the bounds of elements for each bucket
 * kthnumBuckets:  array to store bucket offset.
 */
template <typename T>
__global__ void generateBucketsandSlopes_distributive 
					(T * pivotsLeft, T * pivotsRight, double * slopes,
                     unsigned int * uniqueBucketsCounts, int numUniqueBuckets,
                     unsigned int * kthnumBuckets, int length, int offset, int numBuckets) {

    int index = blockIdx.x * blockDim.x + threadIdx.x;

    // Assign bucket number and slope to first to the second to last active buckets
    if (index < numUniqueBuckets - 1) {
        for (int i = index; i < numUniqueBuckets - 1; i += offset) {

            // assign bucket number
            kthnumBuckets[i] = max(uniqueBucketsCounts[i] * numBuckets / length, 2);

            // assign slope
            slopes[i] = (double) kthnumBuckets[i] / (double) (pivotsRight[i] - pivotsLeft[i]);

            if (isinf(slopes[i]))
                slopes[i] = 0;
        }
    }

    // Assign bucket number and slope to the last active buckets
    if (index < 1) {
        // exclusive cumulative sum to the kthnumbuckets for finding the correct number of buckets
        // for the last active buckets
        thrust::exclusive_scan(thrust::device, kthnumBuckets, 
        					   kthnumBuckets + numUniqueBuckets, kthnumBuckets, 0);


        // assign slope
        slopes[numUniqueBuckets - 1] = (numBuckets - kthnumBuckets[numUniqueBuckets - 1])
                                       / (double) (pivotsRight[numUniqueBuckets - 1] - 
                                       			   pivotsLeft[numUniqueBuckets - 1]);

        if (isinf(slopes[numUniqueBuckets - 1]))
            slopes[numUniqueBuckets - 1] = 0;
    }
    
    __syncthreads();
    
    // if we have extreme cases
    if (kthnumBuckets[numUniqueBuckets - 1] >= numBuckets) {
    	if (index < numUniqueBuckets - 1) {
	        for (int i = index; i < numUniqueBuckets - 1; i += offset) {

	            // assign bucket number
	            kthnumBuckets[i] = max(uniqueBucketsCounts[i] * numBuckets / length, 1);

	            // assign slope
	            slopes[i] = (double) kthnumBuckets[i] / (double) (pivotsRight[i] - pivotsLeft[i]);

	            if (isinf(slopes[i]))
	                slopes[i] = 0;
	        }
	    }

	    // Assign bucket number and slope to the last active buckets
	    if (index < 1) {
	        // exclusive cumulative sum to the kthnumbuckets for finding the correct number of buckets
	        // for the last active buckets
	        thrust::exclusive_scan(thrust::device, kthnumBuckets, kthnumBuckets + numUniqueBuckets, kthnumBuckets, 0);


	        // assign slope
	        slopes[numUniqueBuckets - 1] = (numBuckets - kthnumBuckets[numUniqueBuckets - 1])
	                                       / (double) (pivotsRight[numUniqueBuckets - 1] - pivotsLeft[numUniqueBuckets - 1]);

	        if (isinf(slopes[numUniqueBuckets - 1]))
	            slopes[numUniqueBuckets - 1] = 0;
	    }
    }
}



/* This function assigns elements to buckets based on the pivots and slopes determined
   by a randomized sampling of the elements in the vector. At the same time, this
   function keeps track of count.

   d_elementToBucket : bucket assignment for every array element
   d_bucketCount : number of element that falls into the indexed buckets within the block
*/
template <typename T>
__global__ void assignSmartBucket_distributive
				(T * d_vector, int length, unsigned int * d_elementToBucket,
                 double * slopes, T * pivotsLeft, T * pivotsRight,
                 unsigned int * kthNumBuckets, unsigned int * d_bucketCount,
                 int numUniqueBuckets, int numBuckets, int offset) {

    int index = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned int bucketIndex;
    int threadIndex = threadIdx.x;

    /*
    if (index == 1) {
        printf("assignmentSmartBucket in kernel\n\n");
    }
     */


    //variables in shared memory for fast access
    extern __shared__ unsigned int array[];
    double * sharedSlopes = (double *)array;
    T * sharedPivotsLeft = (T *)&sharedSlopes[numUniqueBuckets];
    unsigned int * sharedkthNumBuckets = (unsigned int *)&sharedPivotsLeft[numUniqueBuckets];
    unsigned int * sharedBuckets = (unsigned int *)&sharedkthNumBuckets[numUniqueBuckets];

    /*
    if (index < 1) {
        printf("executed to phase 0\n\n");
        /*
        for (int bb=0; bb < 16; bb++) {
            printf("bb=%d, vec=%d, elemtobuck=%d, slopes=%lf buckCout=%d, pleft=%d, pright=%d \n ", 
            	   bb, d_vector[bb], d_elementToBucket[bb],slopes[bb], 
            	   d_bucketCount[bb], pivotsLeft[bb], pivotsRight[bb]);
        }

        printf("\n \n \n");

    }
    */

    //reading bucket counts into shared memory where increments will be performed
    for (int i = 0; i < (numBuckets / MAX_THREADS_PER_BLOCK); i++) {

        if (threadIndex < numBuckets)
            sharedBuckets[i * MAX_THREADS_PER_BLOCK + threadIndex] = 0;
    }

    /*
    //    if (index < length) {
    if (index < 1)
        printf("executed to phase 1\n\n");
        */


    if (threadIndex < numUniqueBuckets) {
        sharedPivotsLeft[threadIndex] = pivotsLeft[threadIndex];
        sharedSlopes[threadIndex] = slopes[threadIndex];
        sharedkthNumBuckets[threadIndex] = kthNumBuckets[threadIndex];
        //printf("PL=%d, Slps=%lf, kNumB=%d \n", pivotsLeft[threadIndex], slopes[threadIndex], kthNumBuckets[threadIndex]);
        //printf("sPL=%d, sSlps=%lf, skNumB=%d \n", sharedPivotsLeft[threadIndex], sharedSlopes[threadIndex], sharedkthNumBuckets[threadIndex]);
    }

    /*
    if (index < 1)
        printf("executed to phase 2\n\n");

    //       if (index < length)
    //         printf("index=%d, length=%d, numUniqueBuckets=%d, offset=%d \n", index, length, numUniqueBuckets, offset);
     */

    __syncthreads();

    /*
    if (index < 1)
        printf("executed to phase 3\n\n");
        */


    //assigning elements to buckets and incrementing the bucket counts
    if (index < length) {
        for (int i = index; i < length; i += offset) {
            T num = d_vector[i];
            int minPivotIndex = 0;
            int maxPivotIndex = numUniqueBuckets;
            int midPivotIndex;

            // find the index of left pivots that is greatest s.t. lower than or equal to
            // num using binary search
            for (int j = 1; j < numUniqueBuckets; j *= 2) {
                midPivotIndex = (maxPivotIndex + minPivotIndex) / 2;
                if (num >= pivotsLeft[midPivotIndex])
                    minPivotIndex = midPivotIndex;
                else
                    maxPivotIndex = midPivotIndex;
            }


            bucketIndex = sharedkthNumBuckets[minPivotIndex]
                          + (unsigned int) (((double)num - (double)sharedPivotsLeft[minPivotIndex])
                                   * sharedSlopes[minPivotIndex]);


            // potential to remove the for loop
            if (sharedPivotsLeft[minPivotIndex] != pivotsRight[minPivotIndex]) {
                if (bucketIndex >= numBuckets) {
                    bucketIndex = numBuckets - 1;
                }
                else if (minPivotIndex < numUniqueBuckets - 1) {
                    if (bucketIndex >= sharedkthNumBuckets[minPivotIndex + 1]) {
                        bucketIndex = sharedkthNumBuckets[minPivotIndex + 1] - 1;
                    }
                }
            }

            d_elementToBucket[i] = bucketIndex;
            atomicInc(sharedBuckets + bucketIndex, length);

            //             printf("%d, %d;  ", d_vector[i], d_elementToBucket[i]);
        }
    }

    /*
    if (index < 1)
        printf("executed to phase 4\n\n");
        */


    //    } // closes the if (index < max(length))

    __syncthreads();

    //reading bucket counts from shared memory back to global memory
    for (int i = 0; i <(numBuckets / MAX_THREADS_PER_BLOCK); i++)
        if (threadIndex < numBuckets)
            *(d_bucketCount + blockIdx.x * numBuckets
              + i * MAX_THREADS_PER_BLOCK + threadIndex) =
                    *(sharedBuckets + i * MAX_THREADS_PER_BLOCK + threadIndex);

    /*
    if (index < 1)
        printf("executed to phase 5\n\n");
        */

} // closes the kernel



/* This function cumulatively sums the count of every block for a given bucket s.t. the
   last block index holds the total number of elements falling into that bucket all over the
   array.
   updates d_bucketCount
*/
__global__ void sumCounts(unsigned int * d_bucketCount, const int numBuckets
        , const int numBlocks) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    for(int j=1; j<numBlocks; j++)
        d_bucketCount[index + numBuckets*j] += d_bucketCount[index + numBuckets*(j-1)];
}



/* This function reindexes the buckets counts for every block according to the
   accumulated d_reindexCounter counter for the reduced vector.
   updates d_bucketCount
*/
__global__ void reindexCounts(unsigned int * d_bucketCount, int numBuckets, int numBlocks,
                              unsigned int * d_reindexCounter, unsigned int * d_uniqueBuckets,
                              const int numUniqueBuckets) {
    int threadIndex = blockIdx.x * blockDim.x + threadIdx.x;

    if (threadIndex < numUniqueBuckets) {
        int index = d_uniqueBuckets[threadIndex];
        unsigned int add = d_reindexCounter[threadIndex];

        for (int j = 0; j < numBlocks; j++)
            d_bucketCount[index + numBuckets * j] += add;
    }
}



/* This function copies the elements of buckets that contain kVals into a newly allocated
   reduced vector space.
   newArray - reduced size vector containing the essential elements
*/
template <typename T>
__global__ void copyElements_distributive 
					(T * d_vector, T * d_newvector, int lengthOld, 
					 unsigned int * elementToBuckets, unsigned int * uniqueBuckets, 
					 int numUniqueBuckets, unsigned int * d_bucketCount, 
					 int numBuckets, unsigned int offset) {

    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int threadIndex;
    int loop = numBuckets / MAX_THREADS_PER_BLOCK;

    extern __shared__ unsigned int sharedBuckets[];

    for (int i = 0; i <= loop; i++) {
        threadIndex = i * blockDim.x + threadIdx.x;
        if (threadIndex < numUniqueBuckets)
            sharedBuckets[threadIndex] = uniqueBuckets[threadIndex];
    }

    __syncthreads();


    if (index < lengthOld) {

        for (int i = index; i < lengthOld; i += offset) {
            unsigned int temp = elementToBuckets[i];


            /*
            for (int j = 0; j < numUniqueBuckets; j++) {
                if (temp == sharedBuckets[j]) {
                    d_newvector[atomicDec(d_bucketCount + blockIdx.x * numBuckets
                                          + sharedBuckets[j], lengthOld) - 1] = d_vector[i];
                    break;
                }
            }
            */



            int minBucketIndex = 0;
            int maxBucketIndex = numUniqueBuckets - 1;
            int midBucketIndex;

            for (int j = 1; j < numUniqueBuckets; j *= 2) {
                midBucketIndex = (maxBucketIndex + minBucketIndex) / 2;
                if (temp > sharedBuckets[midBucketIndex])
                    minBucketIndex = midBucketIndex + 1;
                else
                    maxBucketIndex = midBucketIndex;
            }

            if (temp == sharedBuckets[maxBucketIndex])
                d_newvector[atomicDec(d_bucketCount + blockIdx.x * numBuckets
                                      + sharedBuckets[maxBucketIndex], lengthOld) - 1] = d_vector[i];

        }
    }

    // needs to swap d_vector with d_newvector
}



/* This function copies the elements of buckets that contain kVals into a newly allocated
   reduced vector space.
   newArray - reduced size vector containing the essential elements
*/
template <typename T>
__global__ void updatePivots_distributive
					(T * d_pivotsLeft, T * d_newPivotsLeft, T * d_newPivotsRight,
                     double * slopes, unsigned int * kthnumBuckets, unsigned int * uniqueBuckets,
                     int numUniqueBuckets, int numUniqueBucketsOld, int offset) {

    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < numUniqueBuckets) {
        for (int i = index; i < numUniqueBuckets; i += offset) {
            unsigned int bucket = uniqueBuckets[i];
            int minBucketIndex = 0;
            int maxBucketIndex = numUniqueBucketsOld;
            int midBucketIndex;


            // perform binary search to find kthNumBucket that is greatest s.t. lower than or equal to the bucket
            for (int j = 1; j < numUniqueBucketsOld; j *= 2) {
                midBucketIndex = (maxBucketIndex + minBucketIndex) / 2;
                if (bucket >= kthnumBuckets[midBucketIndex])
                    minBucketIndex = midBucketIndex;
                else
                    maxBucketIndex = midBucketIndex;
            }


            long double invslope=0.0;
            if (slopes[minBucketIndex] == (double)0) {
                d_newPivotsLeft[i] = d_pivotsLeft[minBucketIndex];
                d_newPivotsRight[i] = d_pivotsLeft[minBucketIndex];
            }
            else {
                invslope = 1/((long double) slopes[minBucketIndex]);
                d_newPivotsLeft[i] = (T)((long double) d_pivotsLeft[minBucketIndex] +
                                        (((long double) (bucket - kthnumBuckets[minBucketIndex])) * invslope)); // / slopes[bucketIndex]));
                d_newPivotsRight[i] = (T) ((long double)d_pivotsLeft[minBucketIndex] +
                                         (((long double) (bucket - kthnumBuckets[minBucketIndex] + 1) * invslope)));
//                                               slopes[bucketIndex]));
            }
        }
    }

    // needs to swap pointers of pivotsLeft with newPivotsLeft, pivotsRight with newPivotsRight
}



/*
 * This function finds the actual element for the kth orderstats by giving the list of buckets
 */
template <typename T>
__global__ void updateOutput_distributive 
						(T * d_vector, unsigned int * d_elementToBucket, int lengthOld, 
						 T * d_tempOutput, unsigned int * d_tempKorderBucket, 
						 int tempKorderLength, int offset){

    int index = blockDim.x * blockIdx.x + threadIdx.x;
    
    /*
    if (index < tempKorderLength) {
    	for (int i = index; i < tempKorderLength; i++) {
    		d_tempOutput[i] = 0;
    	}
    }
    */

    if (index < lengthOld) {
        for (int i = index; i < lengthOld; i += offset) {
            unsigned int bucket = d_elementToBucket[i];

            for (int j = 0; j < tempKorderLength; j++) {
                if (d_tempKorderBucket[j] == bucket)
                    d_tempOutput[j] = d_vector[i];
            }
        }
    }
}
                                            
                                            
                                            
                                            
/// ***********************************************************
/// ***********************************************************
/// **** HELPER GPU FUNCTIONS LIBRARIES
/// ***********************************************************
/// ***********************************************************
template <typename T>
void generateSamples_distributive_CALL
			(T* d_vector, T* d_sampleVector, int length_local, int sampleSize_local, int offset) {
	 generateSamples_distributive
	 	<<<sampleSize_local / MAX_THREADS_PER_BLOCK, MAX_THREADS_PER_BLOCK>>> 
	 		(d_vector, d_sampleVector, length_local, sampleSize_local, offset);		
			}
template void generateSamples_distributive_CALL
			(int* d_vector, int* d_sampleVector, 
			 int length_local, int sampleSize_local, int offset);
template void generateSamples_distributive_CALL
			(unsigned int* d_vector, unsigned int* d_sampleVector, 
			 int length_local, int sampleSize_local, int offset);
template void generateSamples_distributive_CALL
			(float* d_vector, float* d_sampleVector, 
			 int length_local, int sampleSize_local, int offset);
template void generateSamples_distributive_CALL
			(double* d_vector, double* d_sampleVector, 
			 int length_local, int sampleSize_local, int offset);


template <typename T>
void generateBucketsandSlopes_distributive_CALL
			(T * pivotsLeft, T * pivotsRight, double * slopes,
             unsigned int * uniqueBucketsCounts, int numUniqueBuckets,
             unsigned int * kthnumBuckets, int length, int offset, 
             int numBuckets, int threadsPerBlock) {
	generateBucketsandSlopes_distributive
		<<<(int) ceil((float)numUniqueBuckets/threadsPerBlock), threadsPerBlock>>>
		        (pivotsLeft, pivotsRight, slopes, uniqueBucketsCounts,
		         numUniqueBuckets, kthnumBuckets, length, offset, numBuckets);
}

template void generateBucketsandSlopes_distributive_CALL
			(int * pivotsLeft, int * pivotsRight, double * slopes,
             unsigned int * uniqueBucketsCounts, int numUniqueBuckets,
             unsigned int * kthnumBuckets, int length, int offset, 
             int numBuckets, int threadsPerBlock);
template void generateBucketsandSlopes_distributive_CALL
			(unsigned int * pivotsLeft, unsigned int * pivotsRight, 
			 double * slopes, unsigned int * uniqueBucketsCounts, 
			 int numUniqueBuckets, unsigned int * kthnumBuckets, 
			 int length, int offset, int numBuckets, int threadsPerBlock);
template void generateBucketsandSlopes_distributive_CALL
			(float * pivotsLeft, float * pivotsRight, double * slopes,
             unsigned int * uniqueBucketsCounts, int numUniqueBuckets,
             unsigned int * kthnumBuckets, int length, int offset, 
             int numBuckets, int threadsPerBlock);
template void generateBucketsandSlopes_distributive_CALL
			(double * pivotsLeft, double * pivotsRight, double * slopes,
             unsigned int * uniqueBucketsCounts, int numUniqueBuckets,
             unsigned int * kthnumBuckets, int length, int offset, 
             int numBuckets, int threadsPerBlock);


template <typename T>
void assignSmartBucket_distributive_CALL
			(T * d_vector, int length, unsigned int * d_elementToBucket,
             double * slopes, T * pivotsLeft, T * pivotsRight,
             unsigned int * kthNumBuckets, unsigned int * d_bucketCount,
             int numUniqueBuckets, int numBuckets, int offset, 
             int numBlocks, int threadsPerBlock) {
                                         
	int sharedMemorySize = numUniqueBuckets * sizeof(T) + 
						   numUniqueBuckets * sizeof(double) + 
                           numUniqueBuckets * sizeof(unsigned int) + 
                           numBuckets * sizeof(unsigned int);
                                         
    assignSmartBucket_distributive<T><<<numBlocks, threadsPerBlock, sharedMemorySize>>>
        		(d_vector, length, d_elementToBucket, slopes, pivotsLeft, pivotsRight,
                 kthNumBuckets, d_bucketCount, numUniqueBuckets, numBuckets, offset);                                
}

template void assignSmartBucket_distributive_CALL
			(int * d_vector, int length, unsigned int * d_elementToBucket,
		     double * slopes, int * pivotsLeft, int * pivotsRight,
		     unsigned int * kthNumBuckets, unsigned int * d_bucketCount,
		     int numUniqueBuckets, int numBuckets, int offset,
		     int numBlocks, int threadsPerBlock);
template void assignSmartBucket_distributive_CALL
			(unsigned * d_vector, int length, unsigned int * d_elementToBucket,
        	 double * slopes, unsigned int * pivotsLeft, unsigned int * pivotsRight,
        	 unsigned int * kthNumBuckets, unsigned int * d_bucketCount,
        	 int numUniqueBuckets, int numBuckets, int offset,
          	 int numBlocks, int threadsPerBlock);
template void assignSmartBucket_distributive_CALL
			(float * d_vector, int length, unsigned int * d_elementToBucket,
        	 double * slopes, float * pivotsLeft, float * pivotsRight,
        	 unsigned int * kthNumBuckets, unsigned int * d_bucketCount,
        	 int numUniqueBuckets, int numBuckets, int offset,
        	 int numBlocks, int threadsPerBlock);
template void assignSmartBucket_distributive_CALL
			(double * d_vector, int length, unsigned int * d_elementToBucket,
        	 double * slopes, double * pivotsLeft, double * pivotsRight,
        	 unsigned int * kthNumBuckets, unsigned int * d_bucketCount,
        	 int numUniqueBuckets, int numBuckets, int offset,
        	 int numBlocks, int threadsPerBlock);
                                            	  


void sumCounts_CALL(unsigned int * d_bucketCount, const int numBuckets, 
					const int numBlocks, int threadsPerBlock) {
	sumCounts<<<numBuckets/threadsPerBlock, threadsPerBlock>>>
		(d_bucketCount, numBuckets, numBlocks);
}



void reindexCounts_CALL(unsigned int * d_bucketCount, int numBuckets, int numBlocks,
                        unsigned int * d_reindexCounter, unsigned int * d_uniqueBuckets,
                        const int numUniqueBuckets, int threadsPerBlock) {
	reindexCounts<<<(int) ceil((float)numUniqueBuckets/threadsPerBlock), threadsPerBlock>>>
          (d_bucketCount, numBuckets, numBlocks, d_reindexCounter, d_uniqueBuckets, 
           numUniqueBuckets);  
                        
}

template <typename T>
void copyElements_distributive_CALL
			(T * d_vector, T * d_newvector, int lengthOld, 
			 unsigned int * elementToBuckets, unsigned int * uniqueBuckets, 
			 int numUniqueBuckets, unsigned int * d_bucketCount, 
			 int numBuckets, unsigned int offset, int threadsPerBlock,
			 int numBlocks) {
	copyElements_distributive<T><<<numBlocks, threadsPerBlock, 
								   numUniqueBuckets * sizeof(unsigned int)>>>
			(d_vector, d_newvector, lengthOld, elementToBuckets, uniqueBuckets, 
			 numUniqueBuckets, d_bucketCount, numBuckets, offset);
}

template void copyElements_distributive_CALL
			(int * d_vector, int * d_newvector, int lengthOld, 
			 unsigned int * elementToBuckets, unsigned int * uniqueBuckets, 
			 int numUniqueBuckets, unsigned int * d_bucketCount, 
			 int numBuckets, unsigned int offset, int threadsPerBlock,
			 int numBlocks);
template void copyElements_distributive_CALL
			(unsigned int * d_vector, unsigned int * d_newvector, int lengthOld, 
			 unsigned int * elementToBuckets, unsigned int * uniqueBuckets, 
			 int numUniqueBuckets, unsigned int * d_bucketCount, 
			 int numBuckets, unsigned int offset, int threadsPerBlock,
			 int numBlocks);
template void copyElements_distributive_CALL
			(float * d_vector, float * d_newvector, int lengthOld, 
			 unsigned int * elementToBuckets, unsigned int * uniqueBuckets, 
			 int numUniqueBuckets, unsigned int * d_bucketCount, 
			 int numBuckets, unsigned int offset, int threadsPerBlock,
			 int numBlocks);
template void copyElements_distributive_CALL
			(double * d_vector, double * d_newvector, int lengthOld, 
			 unsigned int * elementToBuckets, unsigned int * uniqueBuckets, 
			 int numUniqueBuckets, unsigned int * d_bucketCount, 
			 int numBuckets, unsigned int offset, int threadsPerBlock,
			 int numBlocks);
			 
			 
			 
template <typename T>
void updatePivots_distributive_CALL
			(T * d_pivotsLeft, T * d_newPivotsLeft, T * d_newPivotsRight,
             double * slopes, unsigned int * kthnumBuckets, unsigned int * uniqueBuckets,
             int numUniqueBuckets, int numUniqueBucketsOld, int offset, 
             int threadsPerBlock) {
	updatePivots_distributive<T>
		<<<(int)ceil((float)numUniqueBuckets/threadsPerBlock), threadsPerBlock>>>
				(d_pivotsLeft, d_newPivotsLeft, d_newPivotsRight,
                 slopes, kthnumBuckets, uniqueBuckets,
                 numUniqueBuckets, numUniqueBucketsOld, offset); 
}

template void updatePivots_distributive_CALL
			(int * d_pivotsLeft, int * d_newPivotsLeft, int * d_newPivotsRight,
             double * slopes, unsigned int * kthnumBuckets, unsigned int * uniqueBuckets,
             int numUniqueBuckets, int numUniqueBucketsOld, int offset, 
             int threadsPerBlocks);
template void updatePivots_distributive_CALL
			(unsigned int * d_pivotsLeft, unsigned int * d_newPivotsLeft, 
			 unsigned int * d_newPivotsRight, double * slopes, unsigned int * kthnumBuckets, 
			 unsigned int * uniqueBuckets, int numUniqueBuckets, int numUniqueBucketsOld, 
			 int offset, int threadsPerBlocks);
template void updatePivots_distributive_CALL
			(float * d_pivotsLeft, float * d_newPivotsLeft, float * d_newPivotsRight,
             double * slopes, unsigned int * kthnumBuckets, unsigned int * uniqueBuckets,
             int numUniqueBuckets, int numUniqueBucketsOld, int offset, 
             int threadsPerBlocks);
template void updatePivots_distributive_CALL
			(double * d_pivotsLeft, double * d_newPivotsLeft, double * d_newPivotsRight,
             double * slopes, unsigned int * kthnumBuckets, unsigned int * uniqueBuckets,
             int numUniqueBuckets, int numUniqueBucketsOld, int offset, 
             int threadsPerBlocks);
			
			

template <typename T>
void updateOutput_distributive_CALL
			(T * d_vector, unsigned int * d_elementToBucket, int lengthOld, 
			 T * d_tempOutput, unsigned int * d_tempKorderBucket, 
			 int tempKorderLength, int offset, int threadsPerBlock) {
	updateOutput_distributive<<<(int)ceil((float)lengthOld/threadsPerBlock), threadsPerBlock>>>
			 (d_vector, d_elementToBucket, lengthOld, d_tempOutput, d_tempKorderBucket, 
			  tempKorderLength, offset);
}
			 
template void updateOutput_distributive_CALL
				(int * d_vector, unsigned int * d_elementToBucket, int lengthOld, 
				 int * d_tempOutput, unsigned int * d_tempKorderBucket, 
				 int tempKorderLength, int offset, int threadsPerBlock);
template void updateOutput_distributive_CALL
				(unsigned int * d_vector, unsigned int * d_elementToBucket, int lengthOld, 
				 unsigned int * d_tempOutput, unsigned int * d_tempKorderBucket, 
				 int tempKorderLength, int offset, int threadsPerBlock);
template void updateOutput_distributive_CALL
				(float * d_vector, unsigned int * d_elementToBucket, int lengthOld, 
				 float * d_tempOutput, unsigned int * d_tempKorderBucket, 
				 int tempKorderLength, int offset, int threadsPerBlock);
template void updateOutput_distributive_CALL
				(double * d_vector, unsigned int * d_elementToBucket, int lengthOld, 
				 double * d_tempOutput, unsigned int * d_tempKorderBucket, 
				 int tempKorderLength, int offset, int threadsPerBlock);


/// ***********************************************************
/// ***********************************************************
/// **** GENERATE KD PIVOTS
/// ***********************************************************
/// ***********************************************************

/* Hash function using Monte Carlo method
 */
__host__ __device__
unsigned int myhash(unsigned int a) {
    a = (a+0x7ed55d16) + (a<<12);
    a = (a^0xc761c23c) ^ (a>>19);
    a = (a+0x165667b1) + (a<<5);
    a = (a+0xd3a2646c) ^ (a<<9);
    a = (a+0xfd7046c5) + (a<<3);
    a = (a^0xb55a4f09) ^ (a>>16);
    return a;
}



/* RandomNumberFunctor
 */
struct RandomNumberFunctor :
        public thrust::unary_function<unsigned int, float> {
    unsigned int mainSeed;

    RandomNumberFunctor(unsigned int _mainSeed) :
            mainSeed(_mainSeed) {}

    __host__ __device__
    float operator()(unsigned int threadIdx)
    {
        unsigned int seed = myhash(threadIdx) * mainSeed;

        thrust::default_random_engine rng(seed);
        rng.discard(threadIdx);
        thrust::uniform_real_distribution<float> u(0, 1);

        return u(rng);
    }
};



/* This function creates a random vector of 1024 elements in the range [0 1]
 */
template <typename T>
void createRandomVector(T * d_vec, int size) {
    timeval t1;
    unsigned int seed;

    gettimeofday(&t1, NULL);
    // seed = t1.tv_usec * t1.tv_sec;
    seed = 1000000000;

    thrust::device_ptr<T> d_ptr(d_vec);
    thrust::transform (thrust::counting_iterator<unsigned int>(0),
                       thrust::counting_iterator<unsigned int>(size),
                       d_ptr, RandomNumberFunctor(seed));
}



/* This function maps the [0 1] range to the [0 vectorSize] and
   grabs the corresponding elements.
*/
template <typename T>
__global__ void enlargeIndexAndGetElements (T * in, T * list, int size) {
    *(in + blockIdx.x*blockDim.x + threadIdx.x) =
            *(list + ((int) (*(in + blockIdx.x * blockDim.x + threadIdx.x) * size)));
}

__global__ void enlargeIndexAndGetElements (float * in, int * out, int * list, int size) {
    *(out + blockIdx.x * blockDim.x + threadIdx.x) =
            (int) *(list + ((int) (*(in + blockIdx.x * blockDim.x + threadIdx.x) * size)));
}


__global__ void enlargeIndexAndGetElements (float * in, unsigned int * out, unsigned int * list, int size) {
    *(out + blockIdx.x * blockDim.x + threadIdx.x) =
            (unsigned int) *(list + ((int) (*(in + blockIdx.x * blockDim.x + threadIdx.x) * size)));
}



/* This function generates Pivots from the random sampled data and calculates slopes.

   pivots - arrays of pivots
   slopes - array of slopes
*/
template <typename T>
void generatePivots (int * pivots, double * slopes, int * d_list, int sizeOfVector
        , int numPivots, int sizeOfSample, int totalSmallBuckets, int min, int max) {

    float * d_randomFloats;
    int * d_randomInts;
    int endOffset = 22;
    int pivotOffset = (sizeOfSample - endOffset * 2) / (numPivots - 3);
    int numSmallBuckets = totalSmallBuckets / (numPivots - 1);

    hipMalloc (&d_randomFloats, sizeof (float) * sizeOfSample);

    d_randomInts = (int *) d_randomFloats;

    createRandomVector (d_randomFloats, sizeOfSample);

    // converts randoms floats into elements from necessary indices
    enlargeIndexAndGetElements<<<(sizeOfSample/MAX_THREADS_PER_BLOCK)
    , MAX_THREADS_PER_BLOCK>>>(d_randomFloats, d_randomInts, d_list,
                               sizeOfVector);



    pivots[0] = min;
    pivots[numPivots-1] = max;

    thrust::device_ptr<T>randoms_ptr(d_randomInts);
    thrust::sort(randoms_ptr, randoms_ptr + sizeOfSample);

    hipDeviceSynchronize();

    // set the pivots which are next to the min and max pivots using the random element
    // endOffset away from the ends
    hipMemcpy (pivots + 1, d_randomInts + endOffset - 1, sizeof (int)
            , hipMemcpyDeviceToHost);
    hipMemcpy (pivots + numPivots - 2, d_randomInts + sizeOfSample - endOffset - 1,
                sizeof (int), hipMemcpyDeviceToHost);
    slopes[0] = numSmallBuckets / (double) (pivots[1] - pivots[0]);

    for (int i = 2; i < numPivots - 2; i++) {
        hipMemcpy (pivots + i, d_randomInts + pivotOffset * (i - 1) + endOffset - 1,
                    sizeof (int), hipMemcpyDeviceToHost);
        slopes[i - 1] = numSmallBuckets / (double) (pivots[i] - pivots[i - 1]);
    }

    // printf("\n\n\n\n%d %d %d %d %d\n\n\n\n", pivots[0], pivots[4], pivots[7], pivots[10], pivots[16]);

    slopes[numPivots - 3] = numSmallBuckets /
                            (double) (pivots[numPivots - 2] - pivots[numPivots - 3]);
    slopes[numPivots - 2] = numSmallBuckets /
                            (double) (pivots[numPivots - 1] - pivots[numPivots - 2]);

    hipFree(d_randomFloats);
}

template void generatePivots<int>(int * pivots, double * slopes, int * d_list, int sizeOfVector, 
							 int numPivots, int sizeOfSample, int totalSmallBuckets, int min, int max);



/* This function generates Pivots from the random sampled data and calculates slopes.

   pivots - arrays of pivots
   slopes - array of slopes
*/
template <typename T>
void generatePivots (unsigned int * pivots, double * slopes, unsigned int * d_list, int sizeOfVector
        , int numPivots, int sizeOfSample, int totalSmallBuckets, unsigned int min, unsigned int max) {

    float * d_randomFloats;
    unsigned int * d_randomInts;
    int endOffset = 22;
    int pivotOffset = (sizeOfSample - endOffset * 2) / (numPivots - 3);
    int numSmallBuckets = totalSmallBuckets / (numPivots - 1);

    hipMalloc (&d_randomFloats, sizeof (float) * sizeOfSample);

    d_randomInts = (unsigned int *) d_randomFloats;

    createRandomVector (d_randomFloats, sizeOfSample);

    // converts randoms floats into elements from necessary indices
    enlargeIndexAndGetElements<<<(sizeOfSample/MAX_THREADS_PER_BLOCK)
    , MAX_THREADS_PER_BLOCK>>>(d_randomFloats, d_randomInts, d_list,
                               sizeOfVector);



    pivots[0] = min;
    pivots[numPivots-1] = max;

    thrust::device_ptr<T>randoms_ptr(d_randomInts);
    thrust::sort(randoms_ptr, randoms_ptr + sizeOfSample);

    hipDeviceSynchronize();

    // set the pivots which are next to the min and max pivots using the random element
    // endOffset away from the ends
    hipMemcpy (pivots + 1, d_randomInts + endOffset - 1, sizeof (unsigned int)
            , hipMemcpyDeviceToHost);
    hipMemcpy (pivots + numPivots - 2, d_randomInts + sizeOfSample - endOffset - 1,
                sizeof (unsigned int), hipMemcpyDeviceToHost);
    slopes[0] = numSmallBuckets / (double) (pivots[1] - pivots[0]);

    for (int i = 2; i < numPivots - 2; i++) {
        hipMemcpy (pivots + i, d_randomInts + pivotOffset * (i - 1) + endOffset - 1,
                    sizeof (unsigned int), hipMemcpyDeviceToHost);
        slopes[i - 1] = numSmallBuckets / (double) (pivots[i] - pivots[i - 1]);
    }

    // printf("\n\n\n\n%d %d %d %d %d\n\n\n\n", pivots[0], pivots[4], pivots[7], pivots[10], pivots[16]);

    slopes[numPivots - 3] = numSmallBuckets /
                            (double) (pivots[numPivots - 2] - pivots[numPivots - 3]);
    slopes[numPivots - 2] = numSmallBuckets /
                            (double) (pivots[numPivots - 1] - pivots[numPivots - 2]);

    hipFree(d_randomFloats);
}

template void generatePivots<unsigned int>(unsigned int * pivots, double * slopes, unsigned int * d_list, int sizeOfVector, 
        				     int numPivots, int sizeOfSample, int totalSmallBuckets, unsigned int min, unsigned int max);

template <typename T>
void generatePivots (T * pivots, double * slopes, T * d_list, int sizeOfVector
        , int numPivots, int sizeOfSample, int totalSmallBuckets, T min, T max) {
    T * d_randoms;
    int endOffset = 22;
    int pivotOffset = (sizeOfSample - endOffset * 2) / (numPivots - 3);
    int numSmallBuckets = totalSmallBuckets / (numPivots - 1);

    hipMalloc (&d_randoms, sizeof (T) * sizeOfSample);

    createRandomVector (d_randoms, sizeOfSample);

    // converts randoms floats into elements from necessary indices
    enlargeIndexAndGetElements<<<(sizeOfSample/MAX_THREADS_PER_BLOCK)
    , MAX_THREADS_PER_BLOCK>>>(d_randoms, d_list, sizeOfVector);

    pivots[0] = min;
    pivots[numPivots - 1] = max;

    thrust::device_ptr<T>randoms_ptr(d_randoms);
    thrust::sort(randoms_ptr, randoms_ptr + sizeOfSample);

    hipDeviceSynchronize();

    // set the pivots which are endOffset away from the min and max pivots
    hipMemcpy (pivots + 1, d_randoms + endOffset - 1, sizeof (T),
                hipMemcpyDeviceToHost);
    hipMemcpy (pivots + numPivots - 2, d_randoms + sizeOfSample - endOffset - 1,
                sizeof (T), hipMemcpyDeviceToHost);
    slopes[0] = numSmallBuckets / ((double)pivots[1] - (double)pivots[0]);

    for (int i = 2; i < numPivots - 2; i++) {
        hipMemcpy (pivots + i, d_randoms + pivotOffset * (i - 1) + endOffset - 1,
                    sizeof (T), hipMemcpyDeviceToHost);
        slopes[i - 1] = numSmallBuckets / ((double) pivots[i] - (double) pivots[i - 1]);
    }

    slopes[numPivots - 3] = numSmallBuckets /
                            ((double)pivots[numPivots - 2] - (double)pivots[numPivots - 3]);
    slopes[numPivots - 2] = numSmallBuckets /
                            ((double)pivots[numPivots - 1] - (double)pivots[numPivots - 2]);

    hipFree(d_randoms);
}


template void generatePivots(float * pivots, double * slopes, float * d_list, int sizeOfVector, 
							 int numPivots, int sizeOfSample, int totalSmallBuckets, float min, float max);
template void generatePivots(double * pivots, double * slopes, double * d_list, int sizeOfVector, 
							 int numPivots, int sizeOfSample, int totalSmallBuckets, double min, double max);
							 
							 
							 
